#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "cuda_header.h"

#include "opencv2/opencv.hpp"
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>


__global__ void RingFilterGPU(unsigned char* ImageData, unsigned char* d_ResultImage, int rows,
                                int cols, int pud, int threshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (
        x > cols - pud - 1 ||
        y > rows - pud - 1 ||
        x < pud + 1 ||
        y < pud + 1
    )
    {
        d_ResultImage[y * cols + x] = (unsigned char)0;
        return;
    }

    float M = 0;
    for(int i = -pud; i != pud+1; i++){
        M += (int)ImageData[ (y-pud)*cols + x + i];
        M += (int)ImageData[ (y+pud)*cols + x + i];
    }

    for(int i = -pud+1; i != pud; i++){
        M += (int)ImageData[ (y+i)*cols + x - pud];
        M += (int)ImageData[ (y+i)*cols + x + pud];
    }

    M = M/( (4*pud + 2) + (4*pud - 4) );

    if(abs(ImageData[y * cols + x] - M) >= threshold){
        d_ResultImage[y * cols + x] = (unsigned char)255;
    }
    else{
        d_ResultImage[y * cols + x] = (unsigned char)0;
    }
}


extern "C" unsigned char* GPUCalc(unsigned char* ImageData, int rows, int cols, int pud, int threshold){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //есть ли в этом смысл?
//    hipStream_t streams[NUM_STREAMS];
//    for (int i = 0; i < NUM_STREAMS; i++) { hipStreamCreate(&streams[i]); }

    unsigned char *dev_Image;
    unsigned char *dev_Result_Image;

    hipMalloc((void**)&dev_Image, sizeof(unsigned char) * cols * rows);
    hipMalloc((void**)&dev_Result_Image, sizeof(unsigned char) * cols * rows);

    hipMemcpy(dev_Image, ImageData, cols*rows * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 gridSize (ceil(cols / (float)THREAD_DIM), ceil(rows / (float)THREAD_DIM));
    dim3 blockSize (THREAD_DIM, THREAD_DIM);

    RingFilterGPU<<<gridSize, blockSize>>>(
                dev_Image,
                dev_Result_Image,
                rows,
                cols,
                pud,
                threshold
            );

    unsigned char* ResultImage = (unsigned char*)malloc(cols * rows * sizeof(unsigned char));
    hipMemcpy(ResultImage, dev_Result_Image, sizeof(unsigned char) * cols * rows, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time of CUDA Work: %3.1f ms\n", elapsedTime);

    hipEventDestroy( start );
    hipEventDestroy( stop  );

//    free(ResultImage);
    hipFree(dev_Result_Image);
    hipFree(dev_Image);

    return ResultImage;
}


