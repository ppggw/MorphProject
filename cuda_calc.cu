#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuda_header.h"
#include "hip/hip_runtime.h"

#include <vector>
#include <iostream>


__device__ int mLock = 0;


__global__ void RingFilterGPU(unsigned char* ImageData, int* vectorX, int* vectorY,
                              int* counter_for_each_block, int * mutex_for_each_block,
                              int* global_counter, float* cache_M, int rows, int cols, int widthOfWindow, int threshold)
{
    __shared__ int block_vectorX[THREAD_DIM * THREAD_DIM];
    __shared__ int block_vectorY[THREAD_DIM * THREAD_DIM];
    __shared__ int im_values[THREAD_DIM * THREAD_DIM];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (
        x > cols - widthOfWindow/2 ||
        y > rows - widthOfWindow/2 ||
        x < widthOfWindow/2 ||
        y < widthOfWindow/2
    )
    {
        return;
    }

    if(threadIdx.x == 0 && threadIdx.y == 0){
        counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] = 0;
        mutex_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] = 0;
    }

    float M = 0;
    for(int i = -widthOfWindow/2; i <= widthOfWindow/2; i++){
        M += (int)ImageData[ (y-widthOfWindow/2)*cols + x + i];
        M += (int)ImageData[ (y+widthOfWindow/2)*cols + x + i];
    }

    for(int i = -widthOfWindow/2; i <= widthOfWindow/2; i++){
        M += (int)ImageData[ (y+i)*cols + x - widthOfWindow/2];
        M += (int)ImageData[ (y+i)*cols + x + widthOfWindow/2];
    }

    M = M/( (2*(widthOfWindow+1) + 2) + (2*(widthOfWindow+1) - 4) );
    cache_M[y*cols + x] = M;

    if(abs(ImageData[y * cols + x] - M) >= threshold){
        bool blocked = true;
        while(blocked) {
            if(0 == atomicCAS(&mutex_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x], 0, 1)){
                block_vectorX[ counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] ] = x;
                block_vectorY[ counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] ] = y;
                im_values[ counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] ] = ImageData[y * cols + x];
                counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] += 1;

                atomicExch(&mutex_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x], 0);
                __threadfence();
                blocked = false;
            }
        }
        __threadfence();
    }

    __syncthreads();
    //дальше пусть каждая стартовая нить блока обработает массив и найдем максимум
    if(threadIdx.x == 0 && threadIdx.y == 0 && counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x] != 0){
        int max = im_values[0];
        int index = 0;
        for(int i = 1; i != counter_for_each_block[blockIdx.y * GRID_SIZE_X + blockIdx.x]; i++){
            if (im_values[i] > max){
                max = im_values[i];
                index = i;
            }
        }

        bool blocked = true;
        while(blocked) {
            if(0 == atomicCAS(&mLock, 0, 1)) {
                if(*global_counter <= VECTOR_INIT_CAPACITY){
                    vectorX[*global_counter] = block_vectorX[index];
                    vectorY[*global_counter] = block_vectorY[index];
                    *global_counter+=1;
                }
                atomicExch(&mLock, 0);
                __threadfence();

                blocked = false;
            }
        }
        __threadfence();
    }
    __syncthreads();
}


__global__ void DispRingFilterGPU(unsigned char* ImageData, int* vectorObjectX, int* vectorObjectY, float* cache_M,
                                  int* counterObject, int* vectorResultX, int* vectorResultY, int* counterFiltered,
                                  int widthOfWindow, float SKO_Porog, int cols, int rows){
    if( (threadIdx.x + threadIdx.y*THREAD_DIM) >= *counterObject){
        return;
    }

    __shared__ int mutex;

    if(threadIdx.x == 0 && threadIdx.y == 0){
        mutex = 0;
    }

    int pointX = vectorObjectX[threadIdx.x + threadIdx.y * THREAD_DIM];
    int pointY = vectorObjectY[threadIdx.x + threadIdx.y * THREAD_DIM];

    if(pointX < widthOfWindow/2 || cols - pointX < widthOfWindow/2 ||
       pointY < widthOfWindow/2 || rows - pointY < widthOfWindow/2){
        return;
    }

    float SumForSKORing = 0;
    for(int l = -widthOfWindow/2; l <= widthOfWindow/2 + 1; l++){
        SumForSKORing += ((int)ImageData[(pointY - widthOfWindow/2) * cols + pointX + l] - cache_M[pointY*cols + pointX]) *
                ((int)ImageData[(pointY - widthOfWindow/2) * cols + pointX + l] - cache_M[pointY*cols + pointX]);

        SumForSKORing += ((int)ImageData[(pointY + widthOfWindow/2) * cols + pointX + l] - cache_M[pointY*cols + pointX]) *
                ((int)ImageData[(pointY + widthOfWindow/2) * cols + pointX + l] - cache_M[pointY*cols + pointX]);
    }

    for(int l = -widthOfWindow/2 + 1; l <= widthOfWindow/2; l++){
        SumForSKORing += ((int)ImageData[(pointY + l) * cols + pointX - widthOfWindow/2] - cache_M[pointY*cols + pointX]) *
                ((int)ImageData[(pointY + l) * cols + pointX - widthOfWindow/2] - cache_M[pointY*cols + pointX]);

        SumForSKORing += ((int)ImageData[(pointY + l) * cols + pointX + widthOfWindow/2] - cache_M[pointY*cols + pointX]) *
                ((int)ImageData[(pointY + l) * cols + pointX + widthOfWindow/2] - cache_M[pointY*cols + pointX]);
    }

    SumForSKORing = SumForSKORing/( (2*(widthOfWindow+1) + 2) + (2*(widthOfWindow+1) - 4));
    SumForSKORing = sqrt(SumForSKORing);

    if((int)ImageData[pointY * cols + pointX] >= (cache_M[pointY*cols + pointX] + SKO_Porog * SumForSKORing) ){
        bool blocked = true;
        while(blocked) {
            if(0 == atomicCAS(&mutex, 0, 1)) {
                vectorResultX[*counterFiltered] = pointX;
                vectorResultY[*counterFiltered] = pointY;
                *counterFiltered+=1;

                atomicExch(&mutex, 0);
                __threadfence();

                blocked = false;
            }
        }
        __threadfence();
    }

    __syncthreads();
}


extern "C" ContForPoints* GPUCalc(unsigned char* ImageData, int rows, int cols, int pud, int threshold,
                                  int widthForFilter, float SKO_Porog){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //есть ли в этом смысл?
//    hipStream_t streams[NUM_STREAMS];
//    for (int i = 0; i < NUM_STREAMS; i++) { hipStreamCreate(&streams[i]); }

    unsigned char *dev_Image;
    int* dev_X;
    int* dev_Y;
    int* dev_CounterGlobal;
    int* dev_CounterForEachBlock;
    int* dev_MutexForEachBlock;
    float* dev_Cache_M;

    int state = 0;

    hipMalloc((void**)&dev_Image, sizeof(unsigned char) * cols * rows);
    hipMalloc((void**)&dev_X, sizeof(int) * VECTOR_INIT_CAPACITY);
    hipMalloc((void**)&dev_Y, sizeof(int) * VECTOR_INIT_CAPACITY);
    hipMalloc((void**)&dev_CounterGlobal, sizeof(int));
    hipMalloc((void**)&dev_CounterForEachBlock, sizeof(int) * GRID_SIZE_Y * GRID_SIZE_X);
    hipMalloc((void**)&dev_MutexForEachBlock, sizeof(int) * GRID_SIZE_Y * GRID_SIZE_X);
    hipMalloc((void**)&dev_Cache_M, sizeof(float) * cols * rows); //кеширование мат ожидания

    hipMemcpy(dev_Image, ImageData, cols*rows * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(dev_CounterGlobal, &state, sizeof(int), hipMemcpyHostToDevice);

    dim3 gridSize (ceil(cols / (float)THREAD_DIM), ceil(rows / (float)THREAD_DIM)); // x = 20 y = 16
    dim3 blockSize (THREAD_DIM, THREAD_DIM);

    RingFilterGPU<<<gridSize, blockSize>>>(
                dev_Image,
                dev_X,
                dev_Y,
                dev_CounterForEachBlock,
                dev_MutexForEachBlock,
                dev_CounterGlobal,
                dev_Cache_M,
                rows,
                cols,
                pud,
                threshold
            );
    hipDeviceSynchronize();

//    int* res_X = (int*)malloc(sizeof(int) * VECTOR_INIT_CAPACITY);
//    int* res_Y = (int*)malloc(sizeof(int) * VECTOR_INIT_CAPACITY);
//    int* res_counter = (int*)malloc(sizeof(int));
//    hipError_t error1 = hipMemcpy(res_X, dev_X, sizeof(int) * VECTOR_INIT_CAPACITY, hipMemcpyDeviceToHost);
//    hipError_t error2 = hipMemcpy(res_Y, dev_Y, sizeof(int) * VECTOR_INIT_CAPACITY, hipMemcpyDeviceToHost);
//    hipError_t error3 = hipMemcpy(res_counter, dev_CounterGlobal, sizeof(int), hipMemcpyDeviceToHost);

    //filterDisp part
    int* dev_X_filtered;
    int* dev_Y_filtered;
    int* dev_CounterFiltered;
    hipMalloc((void**)&dev_X_filtered, sizeof(int) * VECTOR_INIT_CAPACITY);
    hipMalloc((void**)&dev_Y_filtered, sizeof(int) * VECTOR_INIT_CAPACITY);
    hipMalloc((void**)&dev_CounterFiltered, sizeof(int));

    hipMemcpy(dev_CounterFiltered, &state, sizeof(int), hipMemcpyHostToDevice);

    DispRingFilterGPU<<<dim3(1,1), blockSize>>>(
                      dev_Image,
                      dev_X,
                      dev_Y,
                      dev_Cache_M,
                      dev_CounterGlobal,
                      dev_X_filtered,
                      dev_Y_filtered,
                      dev_CounterFiltered,
                      widthForFilter,
                      SKO_Porog,
                      cols,
                      rows
                  );
    hipDeviceSynchronize();

    int* res_X = (int*)malloc(sizeof(int) * VECTOR_INIT_CAPACITY);
    int* res_Y = (int*)malloc(sizeof(int) * VECTOR_INIT_CAPACITY);
    int* res_counter = (int*)malloc(sizeof(int));
    hipError_t error1 = hipMemcpy(res_X, dev_X_filtered, sizeof(int) * VECTOR_INIT_CAPACITY, hipMemcpyDeviceToHost);
    hipError_t error2 = hipMemcpy(res_Y, dev_Y_filtered, sizeof(int) * VECTOR_INIT_CAPACITY, hipMemcpyDeviceToHost);
    hipError_t error3 = hipMemcpy(res_counter, dev_CounterFiltered, sizeof(int), hipMemcpyDeviceToHost);

    ContForPoints* cont = (ContForPoints*)malloc(sizeof(ContForPoints));
    cont->vectorX = res_X;
    cont->vectorY = res_Y;
    cont->counter = res_counter;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
//    printf("Time of CUDA Work: %3.1f ms\n", elapsedTime);

    hipEventDestroy( start );
    hipEventDestroy( stop  );

    hipFree(dev_Image);
    hipFree(dev_X);
    hipFree(dev_Y);
    hipFree(dev_CounterGlobal);
    hipFree(dev_CounterForEachBlock);
    hipFree(dev_MutexForEachBlock);
    hipFree(dev_Cache_M);

    hipFree(dev_X_filtered);
    hipFree(dev_Y_filtered);
    hipFree(dev_CounterFiltered);

    return cont;
}
